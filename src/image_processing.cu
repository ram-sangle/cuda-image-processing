#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>

__global__ void rgb2gray_kernel(unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int idx = (y * width + x) * 3;
        unsigned char r = input[idx];
        unsigned char g = input[idx + 1];
        unsigned char b = input[idx + 2];
        output[y * width + x] = static_cast<unsigned char>(0.299f * r + 0.587f * g + 0.114f * b);
    }
}

__global__ void sobel_kernel(unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
        int idx = y * width + x;
        int gx = -input[(y - 1) * width + (x - 1)] - 2 * input[y * width + (x - 1)] - input[(y + 1) * width + (x - 1)]
               + input[(y - 1) * width + (x + 1)] + 2 * input[y * width + (x + 1)] + input[(y + 1) * width + (x + 1)];
        int gy = -input[(y - 1) * width + (x - 1)] - 2 * input[(y - 1) * width + x] - input[(y - 1) * width + (x + 1)]
               + input[(y + 1) * width + (x - 1)] + 2 * input[(y + 1) * width + x] + input[(y + 1) * width + (x + 1)];
        output[idx] = min(255, abs(gx) + abs(gy));
    }
}

int main() {
    std::string input_path = "input_images/eiffle-tower.bmp";
    std::string gray_path = "images/gray-eiffle-tower.bmp";
    std::string edge_path = "images/edge-eiffle-tower.bmp";

    cv::Mat input_img = cv::imread(input_path, cv::IMREAD_COLOR);
    int width = input_img.cols;
    int height = input_img.rows;

    unsigned char *d_input, *d_gray, *d_edge;
    size_t rgb_size = width * height * 3;
    size_t gray_size = width * height;

    hipMalloc(&d_input, rgb_size);
    hipMalloc(&d_gray, gray_size);
    hipMalloc(&d_edge, gray_size);

    hipMemcpy(d_input, input_img.data, rgb_size, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    rgb2gray_kernel<<<grid, block>>>(d_input, d_gray, width, height);
    sobel_kernel<<<grid, block>>>(d_gray, d_edge, width, height);

    cv::Mat gray_img(height, width, CV_8UC1);
    cv::Mat edge_img(height, width, CV_8UC1);
    hipMemcpy(gray_img.data, d_gray, gray_size, hipMemcpyDeviceToHost);
    hipMemcpy(edge_img.data, d_edge, gray_size, hipMemcpyDeviceToHost);

    cv::imwrite(gray_path, gray_img);
    cv::imwrite(edge_path, edge_img);

    hipFree(d_input);
    hipFree(d_gray);
    hipFree(d_edge);

    std::cout << "Image processing completed using CUDA." << std::endl;
    return 0;
}
